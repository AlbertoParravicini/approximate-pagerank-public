#include "hip/hip_runtime.h"
// Created by Francesco Sgherzi on 15/04/19.
//

#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <chrono>
#include <fstream>
#include <map>
#include <vector>
#include <algorithm>

#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include "../common/csc_matrix/csc_matrix.h"
#include "../common/utils/utils.hpp"

#define TAU 0.0
#define ALPHA 0.85

#define MAX_B 1024
#define MAX_T 1024

#define DEBUG true

#define USE_NO_OPTIMIZATION false
#define USE_L2_NORM true
#define USE_L2_NORM_BITMASK false
#define GRAPH_TYPE ((std::string) "smw")

#define PYTHON_PAGERANK_VALUES false 
#define PYTHON_CONVERGENCE_ERROR_OUT false

#define MAX_ITER 200

#define num_type long long unsigned

// 0.000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000
#define SCALE 63

__host__
__device__
__forceinline__
num_type d_to_fixed(double x) {
    return x * ((double) ((num_type) 1 << SCALE));
}

__host__
__device__
__forceinline__
num_type fixed_mult(num_type x, num_type y) {
    return d_to_fixed(((double) ((double) x / (double) (((num_type) 1) << SCALE)) * ((double) y / (double) (((num_type) 1) << SCALE))));
}


csc_fixed_t to_fixed_csc(csc_t m) {

    csc_fixed_t fixed_csc;

    fixed_csc.col_idx = m.col_idx;
    fixed_csc.col_ptr = m.col_ptr;
    fixed_csc.col_val = std::vector<num_type>();

    for (int i = 0; i < m.col_val.size(); ++i) {
        fixed_csc.col_val.push_back(d_to_fixed(m.col_val[i]));
    }

    return fixed_csc;

}

template<typename T>
void to_device_csc(T *csc_col_val, int *csc_col_ptr, int *csc_col_idx, const csc_fixed_t src) {

    hipMemcpy(csc_col_val, &src.col_val[0], sizeof(T) * src.col_val.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_col_ptr, &src.col_ptr[0], sizeof(int) * src.col_ptr.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_col_idx, &src.col_idx[0], sizeof(int) * src.col_idx.size(), hipMemcpyHostToDevice);

}

__global__
void d_fixed_set_dangling_bitmap(bool *dangling_bitmap, int *csc_col_idx, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        dangling_bitmap[csc_col_idx[i]] = 0;
    }

}


template<typename T>
__global__
void d_fixed_spmv(T *Y, T *pr, T *csc_col_val, int *csc_col_ptr, int *csc_col_idx, const int DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for (int i = init; i < DIMV; i += stride) {

        int begin = csc_col_ptr[i];
        int end = csc_col_ptr[i + 1];

        T acc = d_to_fixed(0.0);

        for (int j = begin; j < end; ++j) {
            acc += fixed_mult(csc_col_val[j], pr[csc_col_idx[j]]);
        }

        Y[i] = acc;
    }
}

template<typename T>
__global__
void
d_update_fixed_spmv(T *Y, T *pr, T *csc_col_val, int *csc_col_ptr, int *csc_col_idx, bool *update_bitmap, const int DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const T initial_zero = d_to_fixed(0.0);


    for (int i = init; i < DIMV && update_bitmap[i]; i += stride) {

        int begin = csc_col_ptr[i];
        int end = csc_col_ptr[i + 1];
        T acc = initial_zero;

        for (int j = begin; j < end; ++j) {
            acc += fixed_mult(csc_col_val[j], pr[csc_col_idx[j]]);
        }

        Y[i] = acc;

    }
}

template<typename T>
__global__
void d_set_value(T *v, const T value, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        v[i] = value;
    }

}

template<typename T>
__global__
void d_fixed_scale(T *v, T value, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        v[i] = fixed_mult(v[i], value);
    }

}

template<typename T>
__global__
void d_fixed_shift(T *v, T value, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        v[i] = v[i] + value;
    }

}

__device__
        __forceinline__

unsigned d_fixed_abs(const unsigned x, const unsigned y) {
    if (x > y) return x - y;
    else return y - x;
}


template<typename T>
__global__
void d_update_fixed_compute_error(T *error, T *v1, T *v2, bool *update_bitmap, const T max_err, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV && update_bitmap[i]; i += stride) {
        error[i] = d_fixed_abs(v1[i], v2[i]);
        update_bitmap[i] = error[i] >= max_err;
    }

}

template<typename T>
__global__
void d_fixed_compute_error(T *error, T *v1, T *v2, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        error[i] = d_fixed_abs(v1[i], v2[i]);
    }

}

template<typename T>
bool check_error(T *e, const T error, const unsigned DIMV) {
    for (int i = 0; i < DIMV; ++i) {
        if (e[i] > error) return false;
    }
    return true;
}


template<typename T>
struct d_fixed_add_functor : public thrust::binary_function<T, T, T> {
    __device__
    T operator()(const T &x, const T &y) const {
        return x + y;
    }
};

template<typename T, typename S>
struct d_fixed_mult_functor : public thrust::binary_function<T, S, T> {
    __device__
    T operator()(const T &x, const S &y) const {
        return fixed_mult(x, y);
    }
};

template<typename T1, typename T2>
T2 d_fixed_dot(T1 *x, T2 *y, size_t n) {

    return thrust::inner_product(
            thrust::device,
            thrust::device_pointer_cast(x),
            thrust::device_pointer_cast(x + n),
            thrust::device_pointer_cast(y),
            0,
            d_fixed_add_functor<T2>(),
            d_fixed_mult_functor<T2, T1>()
    );
}

template<typename T>
void debug_print(char *name, T *v, const unsigned DIMV) {

    T *test;
    hipHostMalloc(&test, DIMV * sizeof(num_type));
    hipMemcpy(test, v, DIMV * sizeof(num_type), hipMemcpyDeviceToHost);

    std::cout << "---------------------DEBUG:" << name << "-------------------" << std::endl;
    for (int i = 0; i < DIMV; ++i) {

        std::cout << test[i] << std::endl;

    }
    std::cout << "------------------END DEBUG:" << name << "-------------------" << std::endl;

}

/**
 * Performs an axpb operation on the x vector inplace
 * @tparam T Numeric type
 * @param x The vector to scale and shift
 * @param a scaling factor
 * @param b shifting factor
 * @return
 */
template<typename T>
__global__
void d_fixed_axpb(T *x, T a, T b, const unsigned DIMV) {
    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        x[i] = fixed_mult(x[i], a) + b;
    }

}

struct is_over_error {
    __device__
    bool operator()(num_type &x) {
        return x > d_to_fixed(TAU);
    }
};

struct d_fixed_square_functor {
    __device__
    num_type operator()(num_type &x) {
        return fixed_mult(x, x);
    }
};

template<typename T>
T euclidean_error(T *error, const unsigned DIMV) {
    return thrust::transform_reduce(
            thrust::device,
            error,
            error + DIMV,
            d_fixed_square_functor(),
            0.0,
            d_fixed_add_functor<T>()
    );
}


int main() {

    /**
     * HOST
     */
    num_type *pr;
    num_type *error;
    num_type *convergence_error_vector;

    /**
     * DEVICE
     */
    num_type *d_pr;
    num_type *d_error;
    num_type *d_spmv_res;
    num_type *d_csc_col_val;
    int *d_csc_col_ptr;
    int *d_csc_col_idx;
    bool *d_dangling_bitmap;
    bool *d_update_bitmap;

    // TODO: remove hardcoded path!
    csc_t csc_matrix = parse_dir("/home/fra/University/HPPS/Approximate-PR/new_ds/" + GRAPH_TYPE, DEBUG);
    csc_fixed_t fixed_csc = to_fixed_csc(csc_matrix);

    const unsigned NON_ZERO = csc_matrix.col_val.size();
    const unsigned DIM = csc_matrix.col_ptr.size() - 1;

    if (DEBUG) {

        std::cout << "\nFEATURES: " << std::endl;
        std::cout << "\tNumber of non zero elements: " << NON_ZERO << std::endl;
        std::cout << "\tNumber of nodes: " << DIM << std::endl;
        std::cout << "\tSparseness: " << (1 - (((double) NON_ZERO) / (DIM * DIM))) * 100 << "%\n" << std::endl;

    }

    hipHostMalloc(&pr, sizeof(num_type) * DIM);
    hipHostMalloc(&error, sizeof(num_type) * DIM);

    if (DEBUG) {
        std::cout << "Initializing device memory" << std::endl;
    }

    // Create device memory
    hipMalloc(&d_csc_col_val, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_csc_col_ptr, sizeof(int) * (DIM + 1));
    hipMalloc(&d_csc_col_idx, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_pr, sizeof(num_type) * DIM);
    hipMalloc(&d_error, sizeof(num_type) * DIM);
    hipMalloc(&d_spmv_res, sizeof(num_type) * DIM);
    hipMalloc(&d_dangling_bitmap, DIM * sizeof(bool));
    hipMalloc(&d_update_bitmap, DIM * sizeof(bool));

    convergence_error_vector = (num_type *) calloc(MAX_ITER, sizeof(num_type));

    // Transform the std::vectors into device vectors
    to_device_csc(d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, fixed_csc);

    if (DEBUG) {
        std::cout << "Initializing PR, Error, dangling bitmap, update bitmap vecors" << std::endl;
    }

    d_set_value << < MAX_B, MAX_T >> > (d_pr, d_to_fixed(1.0 / DIM), DIM);
    d_set_value << < MAX_B, MAX_T >> > (d_error, d_to_fixed(1.0), DIM);
    d_set_value << < MAX_B, MAX_T >> > (d_dangling_bitmap, true, DIM);
    d_set_value << < MAX_B, MAX_T >> > (d_update_bitmap, true, DIM);

    d_fixed_set_dangling_bitmap << < MAX_B, MAX_T >> > (d_dangling_bitmap, d_csc_col_idx, NON_ZERO);

    // debug_print("d_dangling_bitmap", d_dangling_bitmap, DIM);

    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);
    hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);

    if (DEBUG) {
        std::cout << "Beginning pagerank" << std::endl;
    }

    int iterations = 0;
    bool converged = false;
    const num_type F_ALPHA = d_to_fixed(ALPHA);
    const num_type F_TAU = d_to_fixed(TAU);
    const num_type F_SHIFT = d_to_fixed((1.0 - ALPHA) / DIM);
    const num_type F_DANGLING_SCALE = d_to_fixed(ALPHA / DIM);

    // Start a timer
    auto pr_clock_start = std::chrono::high_resolution_clock::now();

    while (!converged && iterations < MAX_ITER) {

        if(USE_NO_OPTIMIZATION){
            // SpMV
            d_fixed_spmv << < MAX_B, MAX_T >> > (d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, DIM);
            // Dangling nodes handler
            num_type res_v = d_fixed_dot(d_pr, d_dangling_bitmap, DIM);
            // aX + b
            d_fixed_axpb << < MAX_T, MAX_B >> >(d_spmv_res, F_ALPHA, ((num_type) F_SHIFT + fixed_mult(F_DANGLING_SCALE, res_v)), DIM);
            // Compute error
            d_fixed_compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, DIM);

            // Swap back the pagerank values
            hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

            // Check for convergence
            converged = thrust::count_if(thrust::device, d_error, d_error + DIM, is_over_error()) == 0;
        }

        if(USE_L2_NORM){
            // SpMV
            d_fixed_spmv << < MAX_B, MAX_T >> > (d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, DIM);
            // Dangling nodes handler
            num_type res_v = d_fixed_dot(d_pr, d_dangling_bitmap, DIM);
            // aX + b
            d_fixed_axpb << < MAX_T, MAX_B >> >(d_spmv_res, F_ALPHA, ((num_type) F_SHIFT + fixed_mult(F_DANGLING_SCALE, res_v)), DIM);
            // Compute error
            d_fixed_compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, DIM);

            // Compute the l2 norm
            num_type error_euc = euclidean_error(d_error, DIM);
            //convergence_error_vector[iterations] = error_euc;

            // Swap back the pagerank values
            hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

            // Check for convergence
            converged = error_euc <= F_TAU;
        }

        if(USE_L2_NORM_BITMASK){
            // SpMV
            d_update_fixed_spmv<< <MAX_B, MAX_T>> > (d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, d_update_bitmap, DIM);
            // Dangling nodes handler
            num_type res_v = d_fixed_dot(d_pr, d_dangling_bitmap, DIM);
            // aX + b
            d_fixed_axpb << < MAX_T, MAX_B >> >(d_spmv_res, F_ALPHA, ((num_type) F_SHIFT + fixed_mult(F_DANGLING_SCALE, res_v)), DIM);
            // Compute error and bitmask
            d_update_fixed_compute_error << <MAX_B, MAX_T>> > (d_error, d_spmv_res, d_pr, d_update_bitmap, F_TAU, DIM);

            // Compute the l2 norm
            num_type error_euc = euclidean_error(d_error, DIM);
            // convergence_error_vector[iterations] = error_euc;

            // Swap back the pagerank values
            hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

            // Check for convergence
            converged = error_euc <= F_TAU;
        }

/*
        // SpMV
        d_fixed_spmv << < MAX_B, MAX_T >> > (d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, DIM);
        //d_update_fixed_spmv<< <MAX_B, MAX_T>> > (d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, d_update_bitmap, DIM);

        // Dangling nodes handler
        num_type res_v = d_fixed_dot(d_pr, d_dangling_bitmap, DIM);
        //num_type res_v = h_fixed_dot(DIM, d_dangling_bitmap, d_pr);
        //std::cout << "Thrust: " << res_v << " <-> Host: " << res_v_h << " -> diff: " << h_s_abs(res_v_h, res_v) << std::endl;

        // aX + b
        d_fixed_axpb << < MAX_T, MAX_B >> >(d_spmv_res, F_ALPHA, ((num_type) F_SHIFT + fixed_mult(F_DANGLING_SCALE, res_v)), DIM);

        // Compute error
        d_fixed_compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, DIM);
        //d_update_fixed_compute_error << <MAX_B, MAX_T>> > (d_error, d_spmv_res, d_pr, d_update_bitmap, F_TAU, DIM);
        num_type error_euc = euclidean_error(d_error, DIM);
        convergence_error_vector[iterations] = error_euc;

        hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

        //converged = thrust::count_if(thrust::device, d_error, d_error + DIM, is_over_error()) == 0;
        converged = error_euc <= F_TAU;*/
        iterations++;

    }
    // Stop the timer
    auto pr_clock_end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(pr_clock_end - pr_clock_start).count();

    if (DEBUG) {
        std::cout << "Pagerank converged after " << duration << " ms" << std::endl;
    }

    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);

    if (DEBUG) {
        std::cout << "Pagerank converged after " << iterations << " iterations" << std::endl;
    }

    std::map<int, num_type> pr_map;
    std::vector<std::pair<int, num_type>> sorted_pr;
    std::vector<int> sorted_pr_idxs;

    for (int i = 0; i < DIM; ++i) {
        sorted_pr.push_back({i, pr[i]});
        pr_map[i] = pr[i];
    }


    std::sort(sorted_pr.begin(), sorted_pr.end(),
              [](const std::pair<int, num_type> &l, const std::pair<int, num_type> &r) {
                  if (l.second != r.second)return l.second > r.second;
                  else return l.first > r.first;
              });

    for (auto const &pair: sorted_pr) {
        sorted_pr_idxs.push_back(pair.first);
        //std::cout << pair.first << "," << pair.second << std::endl;
    }
    if (DEBUG) {
        std::cout << "Checking results..." << std::endl;

        std::ifstream results;
        // TODO: remove hardcoded path!
        results.open("/home/fra/University/HPPS/Approximate-PR/new_ds/" + GRAPH_TYPE + "/results.txt");

        int i = 0;
        int tmp = 0;
        int errors = 0;

        int prev_left_idx = 0;
        int prev_right_idx = 0;

        while (results >> tmp) {
            if (tmp != sorted_pr_idxs[i]) {
                if (prev_left_idx != sorted_pr_idxs[i] || prev_right_idx != tmp) {
                    errors++;
                    if (errors <= 10) {
                        // Print only the top 10 errors
                        std::cout << "ERROR AT INDEX " << i << ": " << tmp << " != " << sorted_pr_idxs[i]
                                  << " Value => " << (num_type) pr_map[sorted_pr_idxs[i]] << std::endl;
                    }
                }

                prev_left_idx = tmp;
                prev_right_idx = sorted_pr_idxs[i];

            }
            i++;
        }

        std::cout << "Percentage of error: " << (((double) errors) / (DIM)) * 100 << "%\n" << std::endl;

        std::cout << "End of computation! Freeing memory..." << std::endl;
    }

    if (PYTHON_CONVERGENCE_ERROR_OUT) {
        for (int i = 0; i < iterations; ++i) {
            std::cout << "(" << i << "," << convergence_error_vector[i] << ")" << std::endl;
        }
    }

    if (PYTHON_PAGERANK_VALUES) {
        for (auto const &pair: sorted_pr) {
            std::cout << pair.first << "," << pair.second << std::endl;
        }
    }

    hipFree(&pr);
    hipFree(&error);
    hipFree(&d_pr);
    hipFree(&d_error);
    hipFree(&d_spmv_res);
    hipFree(&d_csc_col_val);
    hipFree(&d_csc_col_ptr);
    hipFree(&d_csc_col_idx);

    hipDeviceReset();

    return 0;
}
