#include "hip/hip_runtime.h"
// Created by Francesco Sgherzi on 15/04/19.
//

#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <chrono>
#include <fstream>
#include <map>
#include <vector>
#include <algorithm>

#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include "../common/csc_matrix/csc_matrix.h"
#include "../common/utils/utils.h"

#define TAU 0.0
#define ALPHA 0.85

#define MAX_B 1024
#define MAX_T 1024

#define MAX_ITER 200

#define num_type float

#define DEBUG true

#define USE_NO_OPTIMIZATION true
#define USE_L2_NORM false
#define USE_L2_NORM_BITMASK false
#define GRAPH_TYPE ((std::string) "smw")

#define PYTHON_PAGERANK_VALUES false
#define PYTHON_CONVERGENCE_ERROR_OUT false

template <typename T>
bool check_error(T *e, const T error, const unsigned DIMV) {
    for (int i = 0; i < DIMV; ++i) {
        if (e[i] > error)
            return false;
    }
    return true;
}

template <typename T>
void to_device_csc(T *csc_col_val, int *csc_col_ptr, int *csc_col_idx, const csc_t src) {

    hipMemcpy(csc_col_val, &src.col_val[0], sizeof(T) * src.col_val.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_col_ptr, &src.col_ptr[0], sizeof(int) * src.col_ptr.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_col_idx, &src.col_idx[0], sizeof(int) * src.col_idx.size(), hipMemcpyHostToDevice);
}

template <typename T>
__global__ void d_set_val(T *m, T value, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {

        for (int i = init; i < DIMV; i += stride) {
            m[i] = value;
        }
    }
}

template <typename T>
__global__ void spmv(T *Y, T *pr, T *csc_col_val, int *csc_col_ptr, int *csc_col_idx, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {

            int begin = csc_col_ptr[i];
            int end = csc_col_ptr[i + 1];

            T acc = 0.0;

            for (int j = begin; j < end; j++) {
                acc += csc_col_val[j] * pr[csc_col_idx[j]];
            }

            Y[i] = acc;
        }
    }
}

template <typename T>
__global__ void part_spmv(T *Y, T *pr, T *csc_col_val, int *csc_col_ptr, int *csc_col_idx, bool *update_bitmap, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV && update_bitmap[i]; i += stride) {

        int begin = csc_col_ptr[i];
        int end = csc_col_ptr[i + 1];
        T acc = 0.0;

        for (int j = begin; j < end; j++) {
            acc += csc_col_val[j] * pr[csc_col_idx[j]];
        }

        Y[i] = acc;
    }
}

template <typename T>
__global__ void scale(T *m, T v, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            m[i] *= v;
        }
    }
}

template <typename T>
__global__ void shift(T *m, T v, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            m[i] += v;
        }
    }
}

/**
 * Performs an axpb operation on the x vector inplace
 * @tparam T Numeric type
 * @param x The vector to scale and shift
 * @param a scaling factor
 * @param b shifting factor
 * @return
 */
template <typename T>
__global__ void axpb(T *x, T a, T b, const unsigned DIMV) {
    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        x[i] = x[i] * a + b;
    }
}

template <typename T>
struct euclidean_functor : public thrust::binary_function<T, T, T> {
    __device__
        T
        operator()(const T &x, const T &y) const {
        return (x - y) * (x - y);
    }
};

// Compute Euclidean norm of the difference of 2 vectors;
template <typename T>
T euclidean_dist(size_t n, T *x, T *y) {
    return std::sqrt(thrust::inner_product(
        thrust::device, thrust::device_pointer_cast(x),
        thrust::device_pointer_cast(x + n), thrust::device_pointer_cast(y),
        0.0f, thrust::plus<T>(), euclidean_functor<T>()));
}

template <typename T>
__global__ void compute_error(T *error, T *next, T *prev, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            error[i] = abs(next[i] - prev[i]);
        }
    }
}

template <typename T>
__global__ void part_compute_error(T *error, T *next, T *prev, bool *update_bitmap, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV && update_bitmap[i]; i += stride) {
        error[i] = abs(next[i] - prev[i]);
        update_bitmap[i] = error[i] >= TAU;
    }
}

__global__ void d_set_dangling_bitmap(bool *dangling_bitmap, int *csc_col_idx, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        dangling_bitmap[csc_col_idx[i]] = 0;
    }
}

template <typename T1, typename T2>
T2 dot(size_t n, T1 *x, T2 *y) {

    return thrust::inner_product(thrust::device_pointer_cast(x),
                                 thrust::device_pointer_cast(x + n),
                                 thrust::device_pointer_cast(y), 0.0);
}

struct is_over_error {
    __device__ bool operator()(num_type &x) {
        return x > TAU;
    }
};

struct d_square : public thrust::unary_function<num_type, num_type> {
    __device__
        num_type
        operator()(num_type &x) {
        return x * x;
    }
};

template <typename T>
T euclidean_error(T *error, const unsigned DIMV) {
    thrust::plus<T> add;
    return thrust::transform_reduce(
        thrust::device,
        error,
        error + DIMV,
        d_square(),
        0.0,
        add);
}

template <typename T1, typename T2>
T2 h_dot(size_t n, T1 *x, T2 *y) {
    T1 *tempx;
    T2 *tempy;

    hipHostMalloc(&tempx, sizeof(T1) * n);
    hipHostMalloc(&tempy, sizeof(T2) * n);

    hipMemcpy(tempx, x, n * sizeof(T1), hipMemcpyDeviceToHost);
    hipMemcpy(tempy, y, n * sizeof(T1), hipMemcpyDeviceToHost);

    T2 acc = 0.0;

    for (int i = 0; i < n; ++i) {
        acc += tempx[i] * tempy[i];
    }

    return acc;
}

int main() {
    hipDeviceReset();

    /**
     * HOST
     */
    num_type *pr;
    num_type *error;
    num_type *convergence_error_vector;

    /**
     * DEVICE
     */
    num_type *d_pr;
    num_type *d_error;
    num_type *d_spmv_res;
    num_type *d_csc_col_val;
    int *d_csc_col_ptr;
    int *d_csc_col_idx;
    bool *d_dangling_bitmap;
    bool *d_update_bitmap;

    // TODO: remove hardcoded path!
    csc_t csc_matrix = parse_dir("/home/fra/University/HPPS/Approximate-PR/new_ds/" + GRAPH_TYPE, DEBUG);

    const unsigned NON_ZERO = csc_matrix.col_val.size();
    const unsigned DIM = csc_matrix.col_ptr.size() - 1;

    if (DEBUG) {
        std::cout << "\nFEATURES: " << std::endl;
        std::cout << "\tNumber of non zero elements: " << NON_ZERO << std::endl;
        std::cout << "\tNumber of nodes: " << DIM << std::endl;
        std::cout << "\tSparseness: " << (1 - (((double)NON_ZERO) / (DIM * DIM))) * 100 << "%\n"
                  << std::endl;
    }

    hipHostMalloc(&pr, sizeof(num_type) * DIM);
    hipHostMalloc(&error, sizeof(num_type) * DIM);

    if (DEBUG) {
        std::cout << "Initializing device memory" << std::endl;
    }

    // Create device memory
    hipMalloc(&d_csc_col_val, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_csc_col_ptr, sizeof(int) * (DIM + 1));
    hipMalloc(&d_csc_col_idx, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_pr, sizeof(num_type) * DIM);
    hipMalloc(&d_error, sizeof(num_type) * DIM);
    hipMalloc(&d_spmv_res, sizeof(num_type) * DIM);
    hipMalloc(&d_dangling_bitmap, DIM * sizeof(bool));
    hipMalloc(&d_update_bitmap, DIM * sizeof(bool));

    convergence_error_vector = (num_type *)calloc(MAX_ITER, sizeof(num_type));

    if (DEBUG) {
        std::cout << "Parsing csc files" << std::endl;
    }

    to_device_csc(d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, csc_matrix);

    if (DEBUG) {
        std::cout << "Initializing pr, error, dangling bitmap vectors" << std::endl;
    }

    // Initialize error and pr vector
    hipMemset(d_pr, (num_type)1.0 / DIM, DIM);
    hipMemset(d_error, (num_type)1.0, DIM);
    hipMemset(d_dangling_bitmap, true, DIM);
    hipMemset(d_update_bitmap, true, DIM);

    d_set_dangling_bitmap<<<MAX_B, MAX_T>>>(d_dangling_bitmap, d_csc_col_idx, NON_ZERO);

    // Copy them back to their host vectors
    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);
    hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);

    if (DEBUG) {
        std::cout << "Beginning pagerank..." << std::endl;
    }

    int iterations = 0;
    bool converged = false;

    auto pr_clock_start = std::chrono::high_resolution_clock::now();

    while (!converged && iterations < MAX_ITER) {

        if (USE_NO_OPTIMIZATION) {
            // SpMV
            spmv<<<MAX_B, MAX_T>>>(d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, DIM);
            // Dangling nodes handler
            num_type res_v = dot(DIM, d_pr, d_dangling_bitmap);
            // aX + b

            axpb<<<MAX_B, MAX_T>>>(
                d_spmv_res,
                (num_type)ALPHA,
                static_cast<num_type>((1.0 - ALPHA) / DIM + (ALPHA / DIM) * res_v),
                DIM);

            // Compute error
            compute_error<<<MAX_B, MAX_T>>>(d_error, d_spmv_res, d_pr, DIM);

            // Swap back the pagerank values
            hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

            // Check for convergence
            converged = thrust::count_if(thrust::device, d_error, d_error + DIM, is_over_error()) == 0;
        }

        if (USE_L2_NORM) {
            // SpMV
            spmv<<<MAX_B, MAX_T>>>(d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, DIM);
            // Dangling nodes handler
            num_type res_v = dot(DIM, d_pr, d_dangling_bitmap);
            // aX + b

            axpb<<<MAX_B, MAX_T>>>(
                d_spmv_res,
                (num_type)ALPHA,
                static_cast<num_type>((1.0 - ALPHA) / DIM + (ALPHA / DIM) * res_v),
                DIM);

            // Compute error
            compute_error<<<MAX_B, MAX_T>>>(d_error, d_spmv_res, d_pr, DIM);

            // Compute the l2 norm
            num_type error_euc = euclidean_error(d_error, DIM);
            convergence_error_vector[iterations] = error_euc;

            // Swap back the pagerank values
            hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

            // Check for convergence
            converged = error_euc <= TAU;
        }

        if (USE_L2_NORM_BITMASK) {
            // SpMV
            part_spmv<<<MAX_B, MAX_T>>>(d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, d_update_bitmap, DIM);
            // Dangling nodes handler
            num_type res_v = dot(DIM, d_pr, d_dangling_bitmap);
            // aX + b

            axpb<<<MAX_B, MAX_T>>>(
                d_spmv_res,
                (num_type)ALPHA,
                static_cast<num_type>((1.0 - ALPHA) / DIM + (ALPHA / DIM) * res_v),
                DIM);

            // Compute error and bitmask
            part_compute_error<<<MAX_B, MAX_T>>>(d_error, d_spmv_res, d_pr, d_update_bitmap, DIM);

            // Compute the l2 norm
            num_type error_euc = euclidean_error(d_error, DIM);
            // convergence_error_vector[iterations] = error_euc;

            // Swap back the pagerank values
            hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

            // Check for convergence
            converged = error_euc <= TAU;
        }
        /*
        spmv << < MAX_B, MAX_T >> > (d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, DIM);
        //part_spmv << < MAX_B, MAX_T >> > (d_spmv_res, d_pr, d_csc_col_val, d_csc_col_ptr, d_csc_col_idx, d_update_bitmap, DIM);

        num_type res_v = dot(DIM, d_dangling_bitmap, d_pr);

        axpb <<< MAX_B, MAX_T >>> (
                d_spmv_res,
                (num_type) ALPHA,
                static_cast<num_type>((1.0 - ALPHA) / DIM + (ALPHA / DIM) * res_v),
                DIM
        );

        //num_type euclidean_error = euclidean_dist(DIM, d_error, d_pr);
        compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, DIM);
        //part_compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, d_update_bitmap, DIM);

        num_type error_euc = euclidean_error(d_error, DIM);
        convergence_error_vector[iterations] = error_euc;
        //std::cout << "Convergence error[" << iterations << "]: " << error_euc << std::endl;

        hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

        //converged = thrust::count_if(thrust::device, d_error, d_error + DIM, is_over_error()) == 0;
        converged = error_euc <= TAU;
        */
        iterations++;
    }

    // Stop the timer
    auto pr_clock_end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(pr_clock_end - pr_clock_start).count();

    if (DEBUG) {
        std::cout << "Pagerank converged after " << duration << " ms" << std::endl;
    }

    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);

    if (DEBUG) {
        std::cout << "converged after n_iter: " << iterations << std::endl;
    }

    std::map<int, num_type> pr_map;
    std::vector<std::pair<int, num_type>> sorted_pr;
    std::vector<int> sorted_pr_idxs;

    for (int i = 0; i < DIM; ++i) {
        sorted_pr.push_back({i, pr[i]});
        pr_map[i] = pr[i];
        //std::cout << "Index: " << i << " => " << pr_map[i] << std::endl;
    }

    std::sort(sorted_pr.begin(), sorted_pr.end(),
              [](const std::pair<int, num_type> &l, const std::pair<int, num_type> &r) {
                  if (l.second != r.second)
                      return l.second > r.second;
                  else
                      return l.first > r.first;
              });

    // print the vector
    for (auto const &pair : sorted_pr) {
        sorted_pr_idxs.push_back(pair.first);
    }

    if (DEBUG) {
        std::cout << "Checking results..." << std::endl;

        std::ifstream results;
        // TODO: remove hardcoded path!
        results.open("/home/fra/University/HPPS/Approximate-PR/new_ds/" + GRAPH_TYPE + "/results.txt");

        int i = 0;
        int tmp = 0;
        int errors = 0;
        int errors_real = 0;

        int prev_left_idx = 0;
        int prev_right_idx = 0;

        while (results >> tmp) {
            // std::cout << "reading " << tmp << std::endl;
            if (tmp != sorted_pr_idxs[i]) {
                errors_real++;
                if (prev_left_idx != sorted_pr_idxs[i] || prev_right_idx != tmp) {
                    errors++;

                    if (errors <= 10) {
                        // Print only the top 10 errors
                        std::cout << "ERROR AT INDEX " << i << ": " << tmp << " != " << sorted_pr_idxs[i]
                                  << " Value => "
                                  << (num_type)pr_map[sorted_pr_idxs[i]] << std::endl;
                    }
                }

                prev_left_idx = tmp;
                prev_right_idx = sorted_pr_idxs[i];
            }
            i++;
        }

        std::cout << "Percentage of error: " << (((double)errors_real) / (DIM)) * 100 << "%\n"
                  << std::endl;
    }

    if (PYTHON_CONVERGENCE_ERROR_OUT) {
        for (int i = 0; i < iterations; ++i) {
            std::cout << "(" << i << "," << convergence_error_vector[i] << ")" << std::endl;
        }
    }

    if (PYTHON_PAGERANK_VALUES) {
        for (auto const &pair : sorted_pr) {
            std::cout << pair.first << "," << pair.second << std::endl;
        }
    }

    hipFree(&pr);
    hipFree(&error);

    hipFree(&d_pr);
    hipFree(&d_error);
    hipFree(&d_spmv_res);
    hipFree(&d_csc_col_val);
    hipFree(&d_csc_col_ptr);
    hipFree(&d_csc_col_idx);

    hipDeviceReset();

    return 0;
}
